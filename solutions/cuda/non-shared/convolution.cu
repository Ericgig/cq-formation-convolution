#include "hip/hip_runtime.h"
//
//  tp2.cpp
//  Exemple de convolution d'image avec lodepng
//
//  Créé par Julien-Charles Lévesque
//  Copyright 2015 Université Laval. Tous droits réservés.
//

#define TILE_WIDTH 2

#include <time.h>
#include "lodepng.h"
#include <iostream>
#include <stdlib.h>
#include <fstream>

#include "Chrono.hpp"
#include "PACC/Tokenizer.hpp"

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(-1);
    }
}

__global__ void convolutionGPU2(int lK, int lWidth, unsigned char *lImage, double *lFilter, unsigned char *outImage)
{
    int fx, fy;
    double lR = 0.;
    double lG = 0.;
    double lB = 0.;
    int x,y;
    int lHalfK = lK/2;
    x = lHalfK + blockDim.x*blockIdx.x + threadIdx.x;
    y = lHalfK + blockDim.y*blockIdx.y + threadIdx.y;
    for (int j = -lHalfK; j <= lHalfK; j++) {
       fy = j + lHalfK;
       for (int i = -lHalfK; i <= lHalfK; i++) {
          fx = i + lHalfK;
          //R[x + i, y + j] = Im[x + i, y + j].R * Filter[i, j]
          lR += double(lImage[(y + j)*lWidth*4 + (x + i)*4    ]) * lFilter[fx + fy*lK];
          lG += double(lImage[(y + j)*lWidth*4 + (x + i)*4 + 1]) * lFilter[fx + fy*lK];
          lB += double(lImage[(y + j)*lWidth*4 + (x + i)*4 + 2]) * lFilter[fx + fy*lK];
       }
    }
    //protection contre la saturation
    if(lR<0.) lR=0.; if(lR>255.) lR=255.;
    if(lG<0.) lG=0.; if(lG>255.) lG=255.;
    if(lB<0.) lB=0.; if(lB>255.) lB=255.;
    //Placer le résultat dans l'image.
    outImage[y*lWidth*4 + x*4] = (unsigned char)lR;
    outImage[y*lWidth*4 + x*4 + 1] = (unsigned char)lG;
    outImage[y*lWidth*4 + x*4 + 2] = (unsigned char)lB;
    outImage[y*lWidth*4 + x*4 + 3] = lImage[y*lWidth*4 + x*4 + 3];
}

using namespace std;

//Aide pour le programme
void usage(char* inName) {
    cout << endl << "Utilisation> " << inName << " fichier_image fichier_noyau [fichier_sortie=output.png]" << endl;
    exit(1);
}

//Décoder à partir du disque dans un vecteur de pixels bruts en un seul appel de fonction
void decode(const char* inFilename,  vector<unsigned char>& outImage, unsigned int& outWidth, unsigned int& outHeight)
{
    //Décoder
    unsigned int lError = lodepng::decode(outImage, outWidth, outHeight, inFilename);

    //Montrer l'erreur s'il y en a une.
    if(lError) 
        cout << "Erreur de décodage " << lError << ": " << lodepng_error_text(lError) << endl;

    //Les pixels sont maintenant dans le vecteur outImage, 4 octets par pixel, organisés RGBARGBA...
}

//Encoder à partir de pixels bruts sur le disque en un seul appel de fonction
//L'argument inImage contient inWidth * inHeight pixels RGBA ou inWidth * inHeight * 4 octets
void encode(const char* inFilename, vector<unsigned char>& inImage, unsigned int inWidth, unsigned int inHeight)
{
    //Encoder l'image
    unsigned lError = lodepng::encode(inFilename, inImage, inWidth, inHeight);

    //Montrer l'erreur s'il y en a une.
    if(lError)
        cout << "Erreur d'encodage " << lError << ": "<< lodepng_error_text(lError) << endl;
}

int main(int inArgc, char *inArgv[])
{
    if(inArgc < 3 or inArgc > 4) usage(inArgv[0]);
    string lFilename = inArgv[1];
    string lOutFilename;
    if (inArgc == 4)
        lOutFilename = inArgv[3];
    else
        lOutFilename = "output.png";

    // Lire le noyau.
    ifstream lConfig;
    lConfig.open(inArgv[2]);
    if (!lConfig.is_open()) {
        cerr << "Le fichier noyau fourni (" << inArgv[2] << ") est invalide." << endl;
        exit(1);
    }
    
    PACC::Tokenizer lTok(lConfig);
    lTok.setDelimiters(" \n","");
        
    string lToken;
    lTok.getNextToken(lToken);
    
    int lK = atoi(lToken.c_str());
    int lHalfK = lK/2;
    
    cout << "Taille du noyau: " <<  lK << endl;
    
    //Lecture du filtre
    double* lFilter = new double[lK*lK];
        
    for (int i = 0; i < lK; i++) {
        for (int j = 0; j < lK; j++) {
            lTok.getNextToken(lToken);
            lFilter[i*lK+j] = atof(lToken.c_str());
        }
    }

    //Lecture de l'image
    //Variables à remplir
    unsigned int lWidth, lHeight; 
    vector<unsigned char> lImage;   //Les pixels bruts
    vector<unsigned char> outImage; //pixels de l'image apres le filtre

    // Variables sur GPU
    unsigned char *dev_lImage;
    double  *dev_lFilter;
    unsigned char *dev_outImage;
//    unsigned char *h_lImage;
//    double  *h_lFilter;
//    unsigned char *h_outImage;

    //Appeler lodepng
    decode(lFilename.c_str(), lImage, lWidth, lHeight);
    outImage.resize((int)lWidth*(int)lHeight*4);
    

    clock_t start, end;
     double cpu_time_used;
     
     start = clock();
    // Allocation de memoire GPU
    hipMalloc((void**)&dev_lImage,sizeof(unsigned char)*lWidth*lHeight*4);
    hipMalloc((void**)&dev_lFilter,sizeof(double)*lK*lK);
    hipMalloc((void**)&dev_outImage,sizeof(unsigned char)*lWidth*lHeight*4);

    checkCUDAError("Malloc failed");

    // Copie de data ver GPU
    hipMemcpy(dev_lImage,&lImage[0],sizeof(unsigned char)*lWidth*lHeight*4,hipMemcpyHostToDevice);
    hipMemcpy(dev_lFilter,lFilter,sizeof(double)*lK*lK,hipMemcpyHostToDevice);
    checkCUDAError("Memcpy failed");
    int numThreadsX = 2;
    int numThreadsY = 15;
    int numBlocksX = ((int)lWidth-lK)/numThreadsX;
    int numBlocksY = ((int)lHeight-lK)/numThreadsY;
    dim3 dimGrid(numBlocksX,numBlocksY);
    dim3 dimBlock(numThreadsX,numThreadsY);

    printf("Lwidth-lK = %d\n",((int)lWidth-lK));
    printf("Lwidth-lK = %d\n",((int)lHeight-lK));
    printf("NumThreadsX= %d\n",numThreadsX);
    printf("NumThreadsY= %d\n",numThreadsY);
    printf("NumBlocksX= %d\n",numBlocksX);
    printf("NumBlocksY= %d\n",numBlocksY);

//    exit(0);
    convolutionGPU2 <<< dimGrid,dimBlock >>> (lK,(int)lWidth,dev_lImage,dev_lFilter,dev_outImage);
    checkCUDAError("convolutionGPU failed");
    
    hipMemcpy(&outImage[0],dev_outImage,sizeof(unsigned char)*lWidth*lHeight*4,hipMemcpyDeviceToHost);
    checkCUDAError("Memcpy back to CPU failed");
    end = clock();
     cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Exec time = %f\n",cpu_time_used);
 
    // The following part of the code can be parallelized the same way.
    //copie les bordures de l'image
    for(int x = 0; x < lHalfK; x++)
    {
        for (int y = 0; y < (int)lHeight; y++)
        {
            outImage[y*lWidth*4 + x*4] = lImage[y*lWidth*4 + x*4];
            outImage[y*lWidth*4 + x*4 + 1] = lImage[y*lWidth*4 + x*4 + 1];
            outImage[y*lWidth*4 + x*4 + 2] = lImage[y*lWidth*4 + x*4 + 2];
            outImage[y*lWidth*4 + x*4 + 3] = lImage[y*lWidth*4 + x*4 + 3];
        }
    }
    for(int x = (int)lWidth-lHalfK; x < (int)lWidth; x++)
    {
        for (int y = 0; y < (int)lHeight; y++)
        {
            outImage[y*lWidth*4 + x*4] = lImage[y*lWidth*4 + x*4];
            outImage[y*lWidth*4 + x*4 + 1] = lImage[y*lWidth*4 + x*4 + 1];
            outImage[y*lWidth*4 + x*4 + 2] = lImage[y*lWidth*4 + x*4 + 2];
            outImage[y*lWidth*4 + x*4 + 3] = lImage[y*lWidth*4 + x*4 + 3];
        }
    }
    for(int x = lHalfK; x < (int)lWidth - lHalfK; x++)
    {
        for (int y = 0; y < lHalfK; y++)
        {
            outImage[y*lWidth*4 + x*4] = lImage[y*lWidth*4 + x*4];
            outImage[y*lWidth*4 + x*4 + 1] = lImage[y*lWidth*4 + x*4 + 1];
            outImage[y*lWidth*4 + x*4 + 2] = lImage[y*lWidth*4 + x*4 + 2];
            outImage[y*lWidth*4 + x*4 + 3] = lImage[y*lWidth*4 + x*4 + 3];
        }
    }
    for(int x = lHalfK; x < (int)lWidth - lHalfK; x++)
    {
        for (int y = (int)lHeight - lHalfK; y < (int)lHeight; y++)
        {
            outImage[y*lWidth*4 + x*4] = lImage[y*lWidth*4 + x*4];
            outImage[y*lWidth*4 + x*4 + 1] = lImage[y*lWidth*4 + x*4 + 1];
            outImage[y*lWidth*4 + x*4 + 2] = lImage[y*lWidth*4 + x*4 + 2];
            outImage[y*lWidth*4 + x*4 + 3] = lImage[y*lWidth*4 + x*4 + 3];
        }
    }
    
    //Sauvegarde de l'image dans un fichier sortie
    encode(lOutFilename.c_str(),  outImage, lWidth, lHeight);

    cout << "L'image a été filtrée et enregistrée dans " << lOutFilename << " avec succès!" << endl;

    delete[] lFilter;
    return 0;
}

